#include "hip/hip_runtime.h"           //CUDA运行时API
#include ""
#include <stdio.h>

int main(){

  hipError_t cudaStatus;
  int num = 0;
  hipDeviceProp_t prop;
  cudaStatus = hipGetDeviceCount(&num);
  for(int i = 0;i<num;i++)
  {
      hipGetDeviceProperties(&prop,i);
  }
  // cudaStatus = addWithCuda(c, a, b, arraySize);

  printf("%s\n", prop.name);
  printf("%d\n", prop.regsPerBlock);
  printf("%d\n", prop.wrapSize);
  printf("%d\n", prop.clockRate)

  return 0;
}
