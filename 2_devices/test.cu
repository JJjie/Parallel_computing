#include "hip/hip_runtime.h"           //CUDA运行时API
#include ""
#include <stdio.h>

int main(){

  hipError_t cudaStatus;
  int num = 0;
  hipDeviceProp_t prop;
  cudaStatus = hipGetDeviceCount(&num);
  for(int i = 0;i<num;i++)
  {
      hipGetDeviceProperties(&prop,i);
  }
  cudaStatus = addWithCuda(c, a, b, arraySize);

  print("%s\n", prop.name);
  print("%d\n", prop.regsPerBlock);
  print("%d\n", prop.wrapSize);
  print("%d\n", prop.clockRate)

  return 0;
}
