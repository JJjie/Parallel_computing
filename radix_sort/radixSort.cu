#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <sys/time.h>


__host__ void cpu_sort(int * const data, const int num_elements){
  int cpu_tmp_0[num_elements];
  int cpu_tmp_1[num_elements];

  for(int bit=0; bit<32; bit++){
    int base_cnt_0 = 0;
    int base_cnt_1 = 0;

    for(int i=0; i<num_elements; i++){
      const int d = data[i];
      const int bit_mask = (1 << bit);
      if ((d & bit_mask) > 0){
        cpu_tmp_1[base_cnt_1] = d;
        base_cnt_1 ++;
      }else{
        cpu_tmp_0[base_cnt_0] = d;
        base_cnt_0 ++;
      }
    }

    for (int i=0; i<base_cnt_0; i++){
      data[i] = cpu_tmp_0[i];
    }

    for(int i=0; i<base_cnt_1; i++){
      data[base_cnt_0 + i] = cpu_tmp_1[i];
    }
  }
}

__global__ void radix_sort(int * const data,
                          const int num_lists, const int num_elements, const int tid,
                          int * const sort_tmp_0, int * const sort_tmp_1){

    for(int bit=0; bit<32; bit++){
      int base_cnt_0 = 0;
      int base_cnt_1 = 0;

      for(int i=0; i<num_elements; i+=num_lists){
        const int d = data[i + tid];
        const int bit_mask = (1 << bit);

        if ((d & bit_mask) > 0){
          sort_tmp_1[base_cnt_1 + tid] = d;
          base_cnt_1 += num_lists;
        }else{
          sort_tmp_0[base_cnt_0 + tid] = d;
          base_cnt_0 += num_lists;
        }
      }

      for (int i=0; i<base_cnt_0; i+=num_lists){
        data[i+tid] = sort_tmp_0[i+tid];
      }

      for(int i=0; i<base_cnt_1; i+=num_lists){
        data[base_cnt_0 + i + tid] = sort_tmp_1[i+tid];
      }
    }
}

hipError_t sort(int * const data,
                const int size){
    int * dev_data = NULL;
    int * sort_tmp_0 = NULL;
    int * sort_tmp_1 = NULL;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&sort_tmp_1, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&sort_tmp_0, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_data, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_data, data, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    radix_sort<<<1, 4>>>(dev_data, 6, size, 0, sort_tmp_0, sort_tmp_1);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(data, dev_data, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
Error:
    hipFree(dev_data);
    hipFree(sort_tmp_0);
    hipFree(sort_tmp_1);
    return cudaStatus;

}

int main(){
  int data[] = {122, 10, 2, 1, 2, 22, 12, 9, 45, 88, 108, 96, 38, 67, 0, 6, 27, 78, 48, 149, 914, 54, 5, 14};
  struct timeval st, et;

  for (unsigned int i = 0; i< 24; i++){
    printf("%d ", data[i]);
  }
  printf("\n");

  // gettimeofday( &st, NULL );
  // cpu_sort(data, 24);
  // gettimeofday( &et, NULL );
  // for (unsigned int i = 0; i< 24; i++){
  //   printf("%d ", data[i]);
  // }
  // printf("\nSerial time: %ld ms\n", (et.tv_sec - st.tv_sec) * 1000 + (et.tv_usec - st.tv_usec)/1000);

  gettimeofday( &st, NULL );
  hipError_t cudaStatus = sort(data, 24);
  if (cudaStatus != hipSuccess)
  {
      fprintf(stderr, "addWithCuda failed!");
      return 1;
  }
  cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess)
  {
      fprintf(stderr, "hipDeviceReset failed!");
      return 1;
  }
  gettimeofday( &et, NULL );
  for (unsigned int i = 0; i< 24; i++){
    printf("%d ", data[i]);
  }
  printf("\nParallel time: %ld ms\n", (et.tv_sec - st.tv_sec) * 1000 + (et.tv_usec - st.tv_usec)/1000);


  return 0;
}
