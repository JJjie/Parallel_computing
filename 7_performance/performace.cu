#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);
__global__ void addKernel_blk(int *c, const int *a, const int *b)
{
    int i = blockIdx.x;
    c[i] = a[i]+ b[i];
}
__global__ void addKernel_thd(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i]+ b[i];
}
int main()
{
    const int arraySize = 1024;
    int a[arraySize] = {0};
    int b[arraySize] = {0};
    for(int i = 0;i<arraySize;i++)
    {
        a[i] = i;
        b[i] = arraySize-i;
    }
    int c[arraySize] = {0};
    // Add vectors in parallel.
    hipError_t cudaStatus;
    int num = 0;
    hipDeviceProp_t prop;
    cudaStatus = hipGetDeviceCount(&num);
    for(int i = 0;i<num;i++)
    {
        hipGetDeviceProperties(&prop,i);
    }
    cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    for(int i = 0;i<arraySize;i++)
    {
        if(c[i] != (a[i]+b[i]))
        {
            printf("Error in %d\n",i);
        }
    }
    return 0;
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    for(int i = 0;i<1000;i++)
    {
//      addKernel_blk<<<size,1>>>(dev_c, dev_a, dev_b);
        addKernel_thd<<<1,size>>>(dev_c, dev_a, dev_b);
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    
    float tm;
    hipEventElapsedTime(&tm,start,stop);
    printf("GPU Elapsed time:%.6f ms.\n",tm);
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return cudaStatus;
}
